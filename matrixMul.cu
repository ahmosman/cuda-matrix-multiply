#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/**
 * Matrix multiplication: C = A * B.
 * Host code.
 *
 * This sample implements matrix multiplication which makes use of shared memory
 * to ensure data reuse, the matrix multiplication is done using tiling approach.
 * It has been written for clarity of exposition to illustrate various CUDA programming
 * principles, not with the goal of providing the most performant generic kernel for matrix multiplication.
 * See also:
 * V. Volkov and J. Demmel, "Benchmarking GPUs to tune dense linear algebra,"
 * in Proc. 2008 ACM/IEEE Conf. on Supercomputing (SC '08),
 * Piscataway, NJ: IEEE Press, 2008, pp. Art. 31:1-11.
 */

/**
 * Changes made to the original code:
 * - Added a parameter to specify how many results each thread computes.
 * - Implemented a templated kernel that can handle different numbers of results per thread.
 * - Adjusted the grid configuration to account for the number of results per thread.
 */

// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

// Helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

// Uniwersalna wersja kernela mnożenia macierzy
// RESULTS_PER_THREAD to parametr określający ile wyników ma obliczać jeden wątek
template <int BLOCK_SIZE, int RESULTS_PER_THREAD>
__global__ void MatrixMulKernel(float *C, float *A, float *B, int wA, int wB)
{
  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  int row = by * BLOCK_SIZE + ty;
  int col = (bx * BLOCK_SIZE + tx) * RESULTS_PER_THREAD; // Każdy wątek liczy RESULTS_PER_THREAD elementów w poziomie

  // Tablica wyników na rejestrach wątku
  float Csub[RESULTS_PER_THREAD] = {0.0f};

  // Pętla po wszystkich kafelkach macierzy A i B
  for (int m = 0; m < wA / BLOCK_SIZE; ++m)
  {
    // Deklaracje pamięci współdzielonej
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE * RESULTS_PER_THREAD]; // x RESULTS_PER_THREAD, bo tyle kolumn potrzebuje każdy wątek

    // Indeksy elementów macierzy A
    int aRow = row;
    int aCol = m * BLOCK_SIZE + tx;

    // Indeks wiersza macierzy B
    int bRow = m * BLOCK_SIZE + ty;

    // Ładujemy dane macierzy A do pamięci współdzielonej
    As[ty][tx] = A[aRow * wA + aCol];

// Ładujemy dane macierzy B do pamięci współdzielonej
// Każdy wątek ładuje RESULTS_PER_THREAD elementów
#pragma unroll
    for (int i = 0; i < RESULTS_PER_THREAD; i++)
    {
      int bCol = col + i;
      Bs[ty][tx * RESULTS_PER_THREAD + i] = B[bRow * wB + bCol];
    }

    __syncthreads();

// Mnożenie macierzy
#pragma unroll
    for (int k = 0; k < BLOCK_SIZE; ++k)
    {
      float aElement = As[ty][k];
#pragma unroll
      for (int i = 0; i < RESULTS_PER_THREAD; i++)
      {
        Csub[i] += aElement * Bs[k][tx * RESULTS_PER_THREAD + i];
      }
    }

    __syncthreads();
  }

// Zapisanie wyników do pamięci globalnej
#pragma unroll
  for (int i = 0; i < RESULTS_PER_THREAD; i++)
  {
    C[row * wB + col + i] = Csub[i];
  }
}

void ConstantInit(float *data, int size, float val)
{
  for (int i = 0; i < size; ++i)
  {
    data[i] = val;
  }
}

// Funkcja testująca mnożenie macierzy dla określonej liczby wyników na wątek
template <int RESULTS_PER_THREAD>
bool RunMatrixMultiplyTest(int block_size, const dim3 &dimsA, const dim3 &dimsB)
{
  printf("\n-------------------------------------------------\n");
  printf("Testowanie mnożenia macierzy z %d wynikami na wątek:\n", RESULTS_PER_THREAD);
  printf("-------------------------------------------------\n");

  // Allocate host memory for matrices A and B
  unsigned int size_A = dimsA.x * dimsA.y;
  unsigned int mem_size_A = sizeof(float) * size_A;
  float *h_A;
  checkCudaErrors(hipHostMalloc(&h_A, mem_size_A));
  unsigned int size_B = dimsB.x * dimsB.y;
  unsigned int mem_size_B = sizeof(float) * size_B;
  float *h_B;
  checkCudaErrors(hipHostMalloc(&h_B, mem_size_B));
  hipStream_t stream;

  // Initialize host memory
  const float valB = 0.01f;
  ConstantInit(h_A, size_A, 1.0f);
  ConstantInit(h_B, size_B, valB);

  // Allocate device memory
  float *d_A, *d_B, *d_C;

  // Allocate host matrix C
  dim3 dimsC(dimsB.x, dimsA.y, 1);
  unsigned int mem_size_C = dimsC.x * dimsC.y * sizeof(float);
  float *h_C;
  checkCudaErrors(hipHostMalloc(&h_C, mem_size_C));

  if (h_C == NULL)
  {
    fprintf(stderr, "Failed to allocate host matrix C!\n");
    return false;
  }

  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_A), mem_size_A));
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_B), mem_size_B));
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_C), mem_size_C));

  // Allocate CUDA events that we'll use for timing
  hipEvent_t start, stop;
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));

  checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

  // copy host memory to device
  checkCudaErrors(
      hipMemcpyAsync(d_A, h_A, mem_size_A, hipMemcpyHostToDevice, stream));
  checkCudaErrors(
      hipMemcpyAsync(d_B, h_B, mem_size_B, hipMemcpyHostToDevice, stream));

  // Setup execution parameters
  dim3 threads(block_size, block_size);

  // Konfiguracja siatki zależna od liczby wyników na wątek
  dim3 grid((dimsB.x + block_size * RESULTS_PER_THREAD - 1) / (block_size * RESULTS_PER_THREAD),
            (dimsA.y + block_size - 1) / block_size);

  printf("Konfiguracja siatki: [%d x %d], wątki/blok: %d\n", grid.x, grid.y, threads.x * threads.y);
  printf("Computing result using CUDA Kernel...\n");

  // Performs warmup operation using MatrixMul CUDA kernel
  if (block_size == 16)
  {
    MatrixMulKernel<16, RESULTS_PER_THREAD><<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
  }
  else
  {
    MatrixMulKernel<32, RESULTS_PER_THREAD><<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
  }

  printf("Wykonano rozgrzewkę\n");
  checkCudaErrors(hipStreamSynchronize(stream));

  // Record the start event
  checkCudaErrors(hipEventRecord(start, stream));

  // Execute the kernel
  int nIter = 1;
  printf("Wykonywanie %d iteracji...\n", nIter);

  for (int j = 0; j < nIter; j++)
  {
    if (block_size == 16)
    {
      MatrixMulKernel<16, RESULTS_PER_THREAD><<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
    }
    else
    {
      MatrixMulKernel<32, RESULTS_PER_THREAD><<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
    }
  }

  // Record the stop event
  checkCudaErrors(hipEventRecord(stop, stream));

  // Wait for the stop event to complete
  checkCudaErrors(hipEventSynchronize(stop));

  float msecTotal = 0.0f;
  checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

  // Compute and print the performance
  float msecPerMatrixMul = msecTotal / nIter;
  double flopsPerMatrixMul = 2.0 * static_cast<double>(dimsA.x) *
                             static_cast<double>(dimsA.y) *
                             static_cast<double>(dimsB.x);
  double gigaFlops =
      (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
  printf(
      "Wydajność dla %d wyników na wątek = %.2f GFlop/s, Czas = %.3f ms, Operacji = %.0f, Wątków/blok = %u\n",
      RESULTS_PER_THREAD, gigaFlops, msecPerMatrixMul, flopsPerMatrixMul, threads.x * threads.y);

  // Copy result from device to host
  checkCudaErrors(
      hipMemcpyAsync(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost, stream));
  checkCudaErrors(hipStreamSynchronize(stream));

  printf("Sprawdzanie poprawności wyników dla %d wyników na wątek: ", RESULTS_PER_THREAD);
  bool correct = true;

  // test relative error by the formula
  //     |<x, y>_cpu - <x,y>_gpu|/<|x|, |y|>  < eps
  double eps = 1.e-4; // machine zero
  int errorCount = 0;
  const int MAX_ERRORS_TO_PRINT = 10;

  for (int i = 0; i < static_cast<int>(dimsC.x * dimsC.y); i++)
  {
    double abs_err = fabs(h_C[i] - (dimsA.x * valB));
    double dot_length = dimsA.x;
    double abs_val = fabs(h_C[i]);
    double rel_err = abs_err / abs_val / dot_length;

    if (rel_err > eps)
    {
      if (errorCount < MAX_ERRORS_TO_PRINT)
      {
        printf("\nBłąd! Matrix[%05d]=%.8f, oczekiwano=%.8f, różnica względna > %E",
               i, h_C[i], dimsA.x * valB, eps);
      }
      errorCount++;
      correct = false;
    }
  }

  if (errorCount > MAX_ERRORS_TO_PRINT)
  {
    printf("\n...i %d więcej błędów", errorCount - MAX_ERRORS_TO_PRINT);
  }

  printf("\nWynik dla %d wyników na wątek = %s\n", RESULTS_PER_THREAD,
         correct ? "POPRAWNY" : "NIEPOPRAWNY");

  // Clean up memory
  checkCudaErrors(hipHostFree(h_A));
  checkCudaErrors(hipHostFree(h_B));
  checkCudaErrors(hipHostFree(h_C));
  checkCudaErrors(hipFree(d_A));
  checkCudaErrors(hipFree(d_B));
  checkCudaErrors(hipFree(d_C));
  checkCudaErrors(hipEventDestroy(start));
  checkCudaErrors(hipEventDestroy(stop));

  return correct;
}

/**
 * Program main - wykonuje testy dla różnych konfiguracji
 */
int main(int argc, char **argv)
{
  printf("[Matrix Multiply Using CUDA] - Starting...\n");

  // This will pick the best possible CUDA capable device
  int dev = findCudaDevice(argc, (const char **)argv);

  int block_size = 32;

  dim3 dimsA(50 * 2 * block_size, 50 * 2 * block_size, 1);
  dim3 dimsB(50 * 2 * block_size, 50 * 2 * block_size, 1);

  if (dimsA.x != dimsB.y)
  {
    printf("Error: outer matrix dimensions must be equal. (%d != %d)\n",
           dimsA.x, dimsB.y);
    exit(EXIT_FAILURE);
  }

  printf("MatrixA(%d,%d), MatrixB(%d,%d)\n", dimsA.x, dimsA.y,
         dimsB.x, dimsB.y);

  checkCudaErrors(hipProfilerStart());

  // Wykonujemy testy dla różnych liczb wyników na wątek
  bool result1 = RunMatrixMultiplyTest<1>(block_size, dimsA, dimsB);
  bool result2 = RunMatrixMultiplyTest<2>(block_size, dimsA, dimsB);
  bool result4 = RunMatrixMultiplyTest<4>(block_size, dimsA, dimsB);
  bool result8 = RunMatrixMultiplyTest<8>(block_size, dimsA, dimsB);

  // Wyświetlenie podsumowania
  printf("\n== PODSUMOWANIE ==\n");
  printf("Test z 1 wynikiem na wątek: %s\n", result1 ? "POPRAWNY" : "NIEPOPRAWNY");
  printf("Test z 2 wynikami na wątek: %s\n", result2 ? "POPRAWNY" : "NIEPOPRAWNY");
  printf("Test z 4 wynikami na wątek: %s\n", result4 ? "POPRAWNY" : "NIEPOPRAWNY");
  printf("Test z 8 wynikami na wątek: %s\n", result8 ? "POPRAWNY" : "NIEPOPRAWNY");

  checkCudaErrors(hipProfilerStop());
  checkCudaErrors(hipDeviceSynchronize());

  if (result1 && result2 && result4 && result8)
    return EXIT_SUCCESS;
  else
    return EXIT_FAILURE;
}