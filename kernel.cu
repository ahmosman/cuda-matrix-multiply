﻿
#include "hip/hip_runtime.h"

#include <time.h>
#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    atomicAdd(&c[i], a[i] + b[i]);
}

int main()
{
    const int arraySize = 1000;
     int a[arraySize];
     int b[arraySize];  
    for (int i = 0; i < arraySize; i++) a[i] = i;
    for (int i = 0; i < arraySize; i++) b[i] = i;

    int c[arraySize] = { 0 };
    

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("wyniki  = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    getchar();
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;
    hipEvent_t start, stop;
    clock_t startt, stopt;
    float msecTotal = 0.0f;
    double gigaFlops = 0;
    
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

   cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    
    cudaStatus = hipEventCreate(&start);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventCreate failed!");
        goto Error;
    }
    cudaStatus = hipEventCreate(&stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventCreate failed!");
        goto Error;
    }

    cudaStatus = hipEventRecord(start, 0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventRecord failed");
        goto Error;
    }
    startt = clock();//jeszcze obliczenia się nie zaczęły

    addKernel<<<1000000, size>>>(dev_c, dev_a, dev_b);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
       
    cudaStatus = hipEventRecord(stop, 0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventRecord failed");
        goto Error;
    }
   
    cudaStatus = hipEventSynchronize(stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cudaEventSynchronize(stop) failed");
        goto Error;
    }
    stopt = clock();//obliczenia już zakończone

    printf("Czas przetwarzania wynosi %.3f msekund\n", ((double)(stopt - startt)));

    
    hipEventElapsedTime(&msecTotal, start, stop);
    //obliczenia predkości obliczeń 
    gigaFlops = 0;//  (liczba operacji arytmetycznych) / (msecTotal / 1000.0f); */
    printf(
        "Performance= %.2f GFlop/s, Time= %.3f msec \n",
        gigaFlops, msecTotal);
   
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
